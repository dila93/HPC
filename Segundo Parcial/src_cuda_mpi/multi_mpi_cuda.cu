#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "extern.h"

__global__ void mult_mat(double *d_a, double *d_b, double *d_c, int height, int width_a, int width_b) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < height && col < width_b) {
    double p_result = 0;
    for (int k = 0; k < width_a; k++) {
      p_result += d_a[row * width_a + k] * d_b[k * width_b + col];
    }
    d_c[row * width_b + col] = p_result;
  }
}

void mult_mat_CUDA(double *h_matriza, double *h_matrizb, double *h_matrizc, int height, int width_a, int width_b) {
  int blocksize = 32;
  double *d_a, *d_b, *d_c;

  // Asign memory in the device
  hipMalloc(&d_a, sizeof(double) * height * width_a);
  hipMalloc(&d_b, sizeof(double) * width_a * width_b);
  hipMalloc(&d_c, sizeof(double) * height * width_b);

  hipMemcpy(d_a, h_matriza, height * width_a * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_matrizb, width_a * width_b * sizeof(double), hipMemcpyHostToDevice);

  dim3 dimBlock(blocksize, blocksize, 1);
  dim3 dimGrid((height / blocksize) + 1, (width_b / blocksize) + 1);

  mult_mat<<< dimGrid, dimBlock >>>(d_a, d_b, d_c, height, width_a, width_b);
  hipMemcpy(h_matrizc, d_c, height * width_b * sizeof(double), hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}

void init_buf(double *matriz, int y, int x) {
  double n = 1;
  for (int i = 0; i < y; i++) {
    for (int j = 0; j < x; j++) {
      matriz[i * x + j] = n++;
    }
  }
}

bool compare_resultado(double *matriz_MPI, double *matriz_CUDA, int y, int x) {
  for (int i = 0; i < y; i++) {
    for (int j = 0; j < x; j++) {
      if (matriz_MPI[i * x + j] != matriz_CUDA[i * x + j])
        return false;
    }
  }
  return true;
}